#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include "aes_cuda.h"
#include "../mpc_keys/uint128_type.h"
#include "../mpc_keys/keys_type.h"
#include "aes_prg_device.h"


__device__ void PRG_cuda(uint32_t *key, uint128_t input, uint128_t& output1, uint128_t& output2, int& bit1, int& bit2){
	input = input.set_lsb_zero();

	uint128_t stash[2];
	stash[0] = input;
	stash[1] = input.reverse_lsb();


    AES_encrypt_cu(stash[0].get_bytes(), stash[0].get_bytes(), key);
    AES_encrypt_cu(stash[1].get_bytes(), stash[1].get_bytes(), key);

	stash[0] = stash[0] ^ input;
	stash[1] = stash[1] ^ input;
	stash[1] = stash[1].reverse_lsb();

	bit1 = stash[0].get_lsb();
	bit2 = stash[1].get_lsb();

	output1 = stash[0].set_lsb_zero();
	output2 = stash[1].set_lsb_zero();
}

__device__ void fss_gen_device(AES_Generator_device* prg, uint32_t *key, uint128_t alpha, int n, DCF_Keys k0, DCF_Keys k1){
	// int maxlayer = n - 7;
	int maxlayer = n;
    const int MAX_LAYER = 64;

	uint128_t s[MAX_LAYER + 1][2];
	int t[MAX_LAYER + 1 ][2];
	uint128_t sCW[MAX_LAYER];
	int tCW[MAX_LAYER][2];

	s[0][0] = prg->random(); 
	s[0][1] = prg->random();
	t[0][0] = s[0][0].get_lsb();
	t[0][1] = t[0][0] ^ 1;
	s[0][0] = s[0][0].set_lsb_zero();
	s[0][1] = s[0][1].set_lsb_zero();

	int i;
	uint128_t s0[2], s1[2]; // 0=L,1=R
	#define LEFT 0
	#define RIGHT 1
	int t0[2], t1[2];
	for(i = 1; i<= maxlayer; i++){
		PRG_cuda(key, s[i-1][0], s0[LEFT], s0[RIGHT], t0[LEFT], t0[RIGHT]);
		PRG_cuda(key, s[i-1][1], s1[LEFT], s1[RIGHT], t1[LEFT], t1[RIGHT]);

		int keep, lose;
        // int alphabit = getbit(alpha, n, i);
        int alphabit = alpha.get_bit(n-i);
		if(alphabit == 0){
			keep = LEFT;
			lose = RIGHT;
		}else{
			keep = RIGHT;
			lose = LEFT;
		}

		sCW[i-1] = s0[lose] ^ s1[lose];

		tCW[i-1][LEFT] = t0[LEFT] ^ t1[LEFT] ^ alphabit ^ 1;
		tCW[i-1][RIGHT] = t0[RIGHT] ^ t1[RIGHT] ^ alphabit;

		if(t[i-1][0] == 1){
			s[i][0] = s0[keep] ^ sCW[i-1];
			t[i][0] = t0[keep] ^ tCW[i-1][keep];
		}else{
			s[i][0] = s0[keep];
			t[i][0] = t0[keep];
		}

		if(t[i-1][1] == 1){
			s[i][1] = s1[keep] ^ sCW[i-1];
			t[i][1] = t1[keep] ^ tCW[i-1][keep];
		}else{
			s[i][1] = s1[keep];
			t[i][1] = t1[keep];
		}
	}

    uint128_t finalblock(0,1);
	finalblock = finalblock ^ s[maxlayer][0];
	finalblock = finalblock ^ s[maxlayer][1];
    // finalblock.print_uint128("finalblock = ", finalblock);

	// unsigned char *buff0;
	// unsigned char *buff1;
	// buff0 = (unsigned char*) malloc(1 + 16 + 1 + 18 * maxlayer + 16);
	// buff1 = (unsigned char*) malloc(1 + 16 + 1 + 18 * maxlayer + 16);

	// if(buff0 == NULL || buff1 == NULL){
	// 	printf("Memory allocation failed\n");
	// 	return;
	// }

	k0[0] = n;
	memcpy(&k0[1], &s[0][0], 16);
	k0[17] = t[0][0];
	for(i = 1; i <= maxlayer; i++){
		memcpy(&k0[18 * i], &sCW[i-1], 16);
		k0[18 * i + 16] = tCW[i-1][0];
		k0[18 * i + 17] = tCW[i-1][1]; 
	}
	memcpy(&k0[18 * maxlayer + 18], &finalblock, 16); 

	k1[0] = n;
	memcpy(&k1[18], &k0[18], 18 * (maxlayer));
	memcpy(&k1[1], &s[0][1], 16);
	k1[17] = t[0][1];
	memcpy(&k1[18 * maxlayer + 18], &finalblock, 16);

	// memcpy(k0, buff0, 1 + 16 + 1 + 18 * maxlayer + 16);
	// memcpy(k1, buff1, 1 + 16 + 1 + 18 * maxlayer + 16);
	// free(buff0);
	// free(buff1);
}

__device__ uint128_t dcf_eval_device(uint32_t *key, DCF_Keys k, uint128_t x){
	int n = k[0];
	int maxlayer = n;
    const int MAX_LAYER = 64;

	uint128_t s[MAX_LAYER + 1];
	int t[MAX_LAYER + 1];
	uint128_t sCW[MAX_LAYER];
	int tCW[MAX_LAYER][2];
	uint128_t finalblock;

	memcpy(&s[0], &k[1], 16);
	t[0] = k[17];

	int i;
	for(i = 1; i <= maxlayer; i++){
		memcpy(&sCW[i-1], &k[18 * i], 16);
		tCW[i-1][0] = k[18 * i + 16];
		tCW[i-1][1] = k[18 * i + 17];
	}

	memcpy(&finalblock, &k[18 * (maxlayer + 1)], 16);

	uint128_t sL, sR;
	uint128_t res(0,0);
	int tL, tR;

    // first layer
    PRG_cuda(key, s[0], sL, sR, tL, tR); 

	sL = sL ^ sCW[0].select(t[0]);
	sR = sR ^ sCW[0].select(t[0]);
	tL = tL ^ (tCW[0][0]*t[0]);
	tR = tR ^ (tCW[0][1]*t[0]);	

	int xbit = x.get_bit(n-1);

    s[1] = sR.select(xbit) ^ sL.select((1-xbit));
    t[1] = tR * xbit + tL * (1-xbit);

    res = res ^ uint128_t(0, xbit*t[0]);

	for(i = 2; i <= maxlayer; i++){
        PRG_cuda(key, s[i - 1], sL, sR, tL, tR); 

		sL = sL ^ sCW[i-1].select(t[i-1]);
		sR = sR ^ sCW[i-1].select(t[i-1]);
		tL = tL ^ (tCW[i-1][0]*t[i-1]);
		tR = tR ^ (tCW[i-1][1]*t[i-1]);	

		int xbit = x.get_bit(n-i);
		s[i] = sR.select(xbit) ^ sL.select((1-xbit));
		t[i] = tR * xbit + tL * (1-xbit);

        int xbit_last = x.get_bit(n-i+1);
        int changed = (xbit_last * (1 - xbit)) | ((1 - xbit_last) * xbit);
        res = res ^ uint128_t(0, changed*t[i-1]);
	}
	xbit = 1-x.get_bit(0);
    res = res ^ uint128_t(0, t[maxlayer]*xbit);
	return res;
}

__global__ void fss_genaeskey_kernel(uint32_t key[4 * (14 + 1)]) {
	// 测试密钥 (16字节 = 128位)
    uint64_t userkey1 = 597349; uint64_t userkey2 = 121379; 
	uint128_t userkey(userkey1, userkey2);
    
    // 扩展密钥
    if (AES_set_encrypt_key_cu(userkey.get_bytes(), 128, key) != 0) {
        printf("Key expansion failed!\n");
        return;
    }
}

__global__ void fss_gen_kernel(uint32_t key[4 * (14 + 1)], uint64_t* alpha, int n,DCF_Keys k0, DCF_Keys k1, int N, int maxlayer) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	uint32_t expanded_key[4 * (14 + 1)];
	memcpy(expanded_key, key, 4 * (14 + 1) * sizeof(uint32_t));
	AES_Generator_device prg;
	unsigned char* k0_local;
	unsigned char* k1_local;
	uint128_t alpha_tid(0, alpha[tid]);
	k0_local = (unsigned char*) (k0 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	k1_local = (unsigned char*) (k1 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	fss_gen_device(&prg, expanded_key, alpha_tid, n, k0_local, k1_local);
}

__global__ void fss_eval_kernel(bool* res, uint32_t key[4 * (14 + 1)], uint64_t* alpha, int n, DCF_Keys k, int N, int maxlayer) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	uint32_t expanded_key[4 * (14 + 1)];
	memcpy(expanded_key, key, 4 * (14 + 1) * sizeof(uint32_t));
	unsigned char* k_local = (unsigned char*) (k + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	uint128_t alpha_tid(0, alpha[tid]);
	res[tid] = dcf_eval_device(expanded_key, k_local, alpha_tid).get_lsb();
}	



__global__ void aes_test_kernel(int N, DCF_Keys k0, DCF_Keys k1) {
    // 测试密钥 (16字节 = 128位)
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	
    uint64_t userkey1 = 597349; uint64_t userkey2 = 121379; 
	uint128_t userkey(userkey1, userkey2);
    
    // 扩展密钥
    uint32_t expanded_key[4 * (14 + 1)];  // AES-128需要11组轮密钥
    if (AES_set_encrypt_key_cu(userkey.get_bytes(), 128, expanded_key) != 0) {
        printf("Key expansion failed!\n");
        return;
    }
    AES_Generator_device prg;
    uint64_t random = prg.random().get_low();
    uint64_t random2 = prg.random().get_low();
    uint128_t output1, output2;

	int maxlayer = 64;
	unsigned char* k0_local;
	unsigned char* k1_local;

	k0_local = (unsigned char*) (k0 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	k1_local = (unsigned char*) (k1 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
    fss_gen_device(&prg, expanded_key, uint128_t(0, random), 64, k0_local, k1_local);

    output1 = dcf_eval_device(expanded_key, k0_local, uint128_t(0, random2));
    output2 = dcf_eval_device(expanded_key, k1_local, uint128_t(0, random2));
    uint128_t res = output1 ^ output2;
    printf("random < random2 = %s\n", (random < random2) == res.get_lsb()? "success" : "failed");
}

__global__ void fss_msb_keygen_kernel(uint32_t key[4 * (14 + 1)], DCF_Keys k0, DCF_Keys k1, int64_t* random0, int64_t* random1, bool* r_msb0, bool* r_msb1, int N, int maxlayer){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	uint32_t expanded_key[4 * (14 + 1)];
	memcpy(expanded_key, key, 4 * (14 + 1) * sizeof(uint32_t));
	AES_Generator_device prg;
	unsigned char* k0_local;
	unsigned char* k1_local;
	k0_local = (unsigned char*) (k0 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	k1_local = (unsigned char*) (k1 + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	uint64_t random0_local = prg.random().get_low();
	uint64_t random1_local = prg.random().get_low();
	uint64_t random_local = random0_local + random1_local;
	// printf("random_local = %lx\n", random_local);
	uint64_t r_prime = ((uint64_t)1 << 63);
	// printf("r_prime = %lx\n", r_prime);
	r_prime = r_prime - (random_local << 1 >> 1);
	// printf("r_prime = %lx\n", r_prime);
	uint128_t random_tid(0, r_prime);
	fss_gen_device(&prg, expanded_key, random_tid, 64, k0_local, k1_local);
	r_msb0[tid] = prg.random().get_lsb();
	r_msb1[tid] = (random_local >> 63) != r_msb0[tid];
	random0[tid] = (int64_t)random0_local;
	random1[tid] = (int64_t)random1_local;
}

__global__ void fss_msb_eval_kernel(bool* res, uint32_t key[4 * (14 + 1)], DCF_Keys k, int64_t* value, bool* r_msb, int N, int maxlayer, int select){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid >= N) return;
	uint32_t expanded_key[4 * (14 + 1)];
	memcpy(expanded_key, key, 4 * (14 + 1) * sizeof(uint32_t));
	unsigned char* k_local = (unsigned char*) (k + tid * (1 + 16 + 1 + 18 * maxlayer + 16));
	// printf("value[tid] = %lx\n", value[tid]);
	uint64_t value_tid = ((uint64_t)value[tid] << 1) >> 1;
	// printf("value_tid = %lx\n", value_tid);
	uint128_t value_tid_128(0, value_tid);
	bool res_local = dcf_eval_device(expanded_key, k_local, value_tid_128).get_lsb();
	// printf("res_local = %d\n", res_local);
	res_local = res_local != r_msb[tid];
	// printf("res_local = %d\n", res_local);
	bool value_msb = ((uint64_t)value[tid]) >> 63;
	// printf("value_msb = %d\n", value_msb);
	res_local = res_local != select*value_msb;
	// printf("res[tid] = %d\n", res_local);
	res[tid] = res_local;
}

extern "C" void cudamsbkeygen(DCF_Keys k0, DCF_Keys k1, int64_t* random0, int64_t* random1, bool* r_msb0, bool* r_msb1, int N, int maxlayer){

	DCF_Keys k0_device;
	hipMalloc(&k0_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));

	DCF_Keys k1_device;
	hipMalloc(&k1_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));

	int64_t* random0_device;
	hipMalloc(&random0_device, N * sizeof(int64_t));
	hipMemcpy(random0_device, random0, N * sizeof(int64_t), hipMemcpyHostToDevice);

	int64_t* random1_device;
	hipMalloc(&random1_device, N * sizeof(int64_t));
	hipMemcpy(random1_device, random1, N * sizeof(int64_t), hipMemcpyHostToDevice);

	bool* r_msb0_device;
	hipMalloc(&r_msb0_device, N * sizeof(bool));
	hipMemcpy(r_msb0_device, r_msb0, N * sizeof(bool), hipMemcpyHostToDevice);

	bool* r_msb1_device;
	hipMalloc(&r_msb1_device, N * sizeof(bool));
	hipMemcpy(r_msb1_device, r_msb1, N * sizeof(bool), hipMemcpyHostToDevice);

	uint32_t* aes_key;
	hipMalloc(&aes_key, 4 * (14 + 1) * sizeof(uint32_t));
	fss_genaeskey_kernel<<<1, 1>>>(aes_key);

	int threads = N > 256 ? 256 : N;
	int blocks = (N + threads - 1) / threads;
	fss_msb_keygen_kernel<<<blocks, threads>>>(aes_key, k0_device, k1_device, random0_device, random1_device, r_msb0_device, r_msb1_device, N, maxlayer);

	hipMemcpy(k0, k0_device, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyDeviceToHost);
	hipMemcpy(k1, k1_device, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyDeviceToHost);
	hipMemcpy(random0, random0_device, N * sizeof(int64_t), hipMemcpyDeviceToHost);
	hipMemcpy(random1, random1_device, N * sizeof(int64_t), hipMemcpyDeviceToHost);
	hipMemcpy(r_msb0, r_msb0_device, N * sizeof(bool), hipMemcpyDeviceToHost);
	hipMemcpy(r_msb1, r_msb1_device, N * sizeof(bool), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
	hipFree(k0_device);
	hipFree(k1_device);
	hipFree(random0_device);
	hipFree(random1_device);
	hipFree(r_msb0_device);
	hipFree(r_msb1_device);
	hipFree(aes_key);
}

extern "C" void cudamsbeval(bool* res, DCF_Keys k, int64_t* value, bool* r_msb, int N, int maxlayer, int party){

	//move MSB_keys to device
	DCF_Keys k_device;
	hipMalloc(&k_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));
	hipMemcpy(k_device, k, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyHostToDevice);

	int64_t* value_device;
	hipMalloc(&value_device, N * sizeof(int64_t));
	hipMemcpy(value_device, value, N * sizeof(int64_t), hipMemcpyHostToDevice);

	bool* r_msb_device;
	hipMalloc(&r_msb_device, N * sizeof(bool));
	hipMemcpy(r_msb_device, r_msb, N * sizeof(bool), hipMemcpyHostToDevice);

	bool* res_device;
	hipMalloc(&res_device, N * sizeof(bool));

	uint32_t* aes_key;
	hipMalloc(&aes_key, 4 * (14 + 1) * sizeof(uint32_t));
	fss_genaeskey_kernel<<<1, 1>>>(aes_key);

	int threads = N > 512 ? 512 : N;
	int blocks = (N + threads - 1) / threads;
	int select = party == 1 ? 1 : 0;

	// hipEvent_t start1, stop1;
    // hipEventCreate(&start1);
    // hipEventCreate(&stop1);

	// hipEventRecord(start1);
	fss_msb_eval_kernel<<<blocks, threads>>>(res_device, aes_key, k_device, value_device, r_msb_device, N, maxlayer, select);
	// hipEventRecord(stop1);

	// hipEventSynchronize(stop1);
	// float time1 = 0;
    // hipEventElapsedTime(&time1, start1, stop1);
    // printf("msb eval Kernel Time taken: %.3f ms\n", time1);

	hipMemcpy(res, res_device, N * sizeof(bool), hipMemcpyDeviceToHost);


	// 等待GPU完成
    hipDeviceSynchronize();
	hipFree(k_device);
	hipFree(value_device);
	hipFree(r_msb_device);
	hipFree(res_device);
	hipFree(aes_key);
}

extern "C" void cudafsskeygen(DCF_Keys k0, DCF_Keys k1, uint64_t* alpha, int N, int n,int maxlayer){

	DCF_Keys k0_device;
	hipMalloc(&k0_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));

	DCF_Keys k1_device;
	hipMalloc(&k1_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));

	uint64_t* alpha_device;
	hipMalloc(&alpha_device, N * sizeof(uint64_t));
	hipMemcpy(alpha_device, alpha, N * sizeof(uint64_t), hipMemcpyHostToDevice);

	uint32_t* aes_key;
	hipMalloc(&aes_key, 4 * (14 + 1) * sizeof(uint32_t));
	fss_genaeskey_kernel<<<1, 1>>>(aes_key);

	int threads = N > 256 ? 256 : N;
	int blocks = (N + threads - 1) / threads;
	fss_gen_kernel<<<blocks, threads>>>(aes_key, alpha_device, n, k0_device, k1_device, N, maxlayer);

	hipMemcpy(k0, k0_device, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyDeviceToHost);
	hipMemcpy(k1, k1_device, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyDeviceToHost);	

	hipFree(k0_device);
	hipFree(k1_device);
	hipFree(alpha_device);
	hipFree(aes_key);
}

extern "C" void cudafsseval(bool *res, DCF_Keys key, uint64_t *value,int N, int maxlayer, int party){

	DCF_Keys key_device;
	hipMalloc(&key_device, N * (1 + 16 + 1 + 18 * maxlayer + 16));
	hipMemcpy(key_device, key, N * (1 + 16 + 1 + 18 * maxlayer + 16), hipMemcpyHostToDevice);

	uint64_t* value_device;
	hipMalloc(&value_device, N * sizeof(uint64_t));
	hipMemcpy(value_device, value, N * sizeof(uint64_t), hipMemcpyHostToDevice);

	bool* res_device;
	hipMalloc(&res_device, N * sizeof(bool));

	uint32_t* aes_key;
	hipMalloc(&aes_key, 4 * (14 + 1) * sizeof(uint32_t));
	fss_genaeskey_kernel<<<1, 1>>>(aes_key);

	int threads = N > 256 ? 256 : N;
	int blocks = (N + threads - 1) / threads;	
    fss_eval_kernel<<<blocks, threads>>>(res_device, aes_key, value_device, 64, key_device, N, maxlayer);   

	hipMemcpy(res, res_device, N * sizeof(bool), hipMemcpyDeviceToHost);

	hipFree(key_device);
	hipFree(value_device);
}

extern "C" int test_dcf() {
    // 启动kernel
	DCF_Keys k0;
	DCF_Keys k1;
	int maxlayer = 64;
	int N = 1000;
	hipMalloc(&k0, N * (1 + 16 + 1 + 18 * maxlayer + 16));
	hipMalloc(&k1, N * (1 + 16 + 1 + 18 * maxlayer + 16));

	bool* res1;
	hipMalloc(&res1, N * sizeof(bool));
	bool* res2;
	hipMalloc(&res2, N * sizeof(bool));

	bool* res1_host;
	hipHostMalloc(&res1_host, N * sizeof(bool));
	bool* res2_host;
	hipHostMalloc(&res2_host, N * sizeof(bool));

	// uint128_t alpha1 = uint128_t(0, 1);
	// uint128_t alpha2 = uint128_t(0, 2);

	uint64_t* alpha1_host;
	hipHostMalloc(&alpha1_host, N * sizeof(uint64_t));
	uint64_t* alpha2_host;
	hipHostMalloc(&alpha2_host, N * sizeof(uint64_t));
	for(int i = 0; i < N; i++){
		alpha1_host[i] = i+1;
		alpha2_host[i] = i+3;
	}

	uint64_t* value1;
	hipMalloc(&value1, N * sizeof(uint64_t));
	uint64_t* value2;
	hipMalloc(&value2, N * sizeof(uint64_t));
	hipMemcpy(value1, alpha1_host, N * sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(value2, alpha2_host, N * sizeof(uint64_t), hipMemcpyHostToDevice);


	int threads = N > 256 ? 256 : N;
	int blocks = (N + threads - 1) / threads;
	uint32_t* aes_key;
	hipMalloc(&aes_key, 4 * (14 + 1) * sizeof(uint32_t));
	fss_genaeskey_kernel<<<1, 1>>>(aes_key);

    // 创建CUDA事件
    hipEvent_t start1, stop1, start2, stop2, start3, stop3;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventCreate(&start3);
    hipEventCreate(&stop3);
    
    // 测量第一个kernel: fss_gen_kernel
    hipEventRecord(start1);
    fss_gen_kernel<<<blocks, threads>>>(aes_key, value1, 64, k0, k1, N, maxlayer);
    hipEventRecord(stop1);
    
    // 测量第二个kernel: first fss_eval_kernel
    hipEventRecord(start2);
    fss_eval_kernel<<<blocks, threads>>>(res1, aes_key, value2, 64, k0, N, maxlayer);
	hipMemcpy(res1_host, res1, N * sizeof(bool), hipMemcpyDeviceToHost);
    hipEventRecord(stop2);
    
    // 测量第三个kernel: second fss_eval_kernel
    hipEventRecord(start3);
    fss_eval_kernel<<<blocks, threads>>>(res2, aes_key, value2, 64, k1, N, maxlayer);
	hipMemcpy(res2_host, res2, N * sizeof(bool), hipMemcpyDeviceToHost);
    hipEventRecord(stop3);
    
    // 同步并获取时间
    hipEventSynchronize(stop1);
    hipEventSynchronize(stop2);
    hipEventSynchronize(stop3);
    
    float time1 = 0, time2 = 0, time3 = 0;
    hipEventElapsedTime(&time1, start1, stop1);
    hipEventElapsedTime(&time2, start2, stop2);
    hipEventElapsedTime(&time3, start3, stop3);
    
    // 打印结果
    printf("fss_gen_kernel time: %.3f ms\n", time1);
    printf("First fss_eval_kernel time: %.3f ms\n", time2);
    printf("Second fss_eval_kernel time: %.3f ms\n", time3);
    
    // 销毁事件
    hipEventDestroy(start1);
    hipEventDestroy(stop1);
    hipEventDestroy(start2);
    hipEventDestroy(stop2);
    hipEventDestroy(start3);
    hipEventDestroy(stop3);
	
	for(int i = 0; i < N; i++){
		bool res = res1_host[i] ^ res2_host[i];
		printf("res = %d\n", res);
	}
    
    
    // 等待GPU完成
    hipDeviceSynchronize();

	hipFree(k0);
	hipFree(k1);
	hipFree(res1);
	hipFree(res2);
		// hipFree(alpha1_host);
		// hipFree(alpha2_host);
    
    // 检查错误
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return -1;
    }

    return 0;
}